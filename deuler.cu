#include "hip/hip_runtime.h"
﻿//  2016 Juncheng E at PIMS.

#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <iomanip>  
#include <ctime>
#include <stdlib.h>
#include <string.h>

using namespace std;

int count = 0;
int nmax; // maximum atom number
int natom; // actual atom number
int numgrain; // the number of grains
int nx,ny,nz; // number of primitive cells in each direction
float a; // lattice constant
float a2; // lattice constant^2
float lx,ly,lz; // the size of the simulation cell
float *alpha, *beta, *gama;
float3 *gr_centerp; // the centers of each of the grains
float ratio = 1.0;
float3 *r; // atom positon
float temp,mass; // temperature, mass
int *atom_grain, *atom_neigh;
bool *atom_id;
//float *atom_mini;
//float *drlist;
int3 DIM;
__constant__ float d_lx, d_ly, d_lz, d_a;
__constant__ int d_nmax, d_numgrain, d_nx[2], d_ny[2], d_nz[2], d_natom, d_GBatoms;


int read_config(char* ifn)
{
	int i;
	ifstream ifile;
	string linebuffer;
	stringstream ss;
	ifile.open(ifn);
	cout << "Read " << ifn << "..." << endl;
	getline(ifile, linebuffer);
	ss << linebuffer;
	ss >> a;
	a2 = a*a;
	ss.str(""); // Clean up ss
	ss.clear(); // Clean up ss
	getline(ifile, linebuffer);
	ss << linebuffer;
	ss >> numgrain >> lx >> ly >> lz;
	ss.str(""); // Clean up ss
	ss.clear(); // Clean up ss
	printf("Number of grains: %d\n",numgrain);
	gr_centerp = new float3 [numgrain];
	alpha = new float [numgrain];
	beta = new float [numgrain];
	gama = new float [numgrain];
	getline(ifile, linebuffer);
	ss << linebuffer;
	ss >> mass >> temp;
	ss.str(""); // Clean up ss
	ss.clear(); // Clean up ss
	for ( i = 0; i < numgrain; ++i )
	{
		getline(ifile, linebuffer);
		ss << linebuffer;
		ss >> gr_centerp[i].x >> gr_centerp[i].y >> gr_centerp[i].z >> alpha[i] >> beta[i] >> gama[i];
//		alpha[i] = alpha[i]*M_PI/180.0;
//		beta[i] = beta[i]*M_PI/180.0;
//		gama[i] = gama[i]*M_PI/180.0;
		ss.str(""); // Clean up ss
		ss.clear(); // Clean up ss
	}
	ifile.close();
	return 0;
}

__device__ float pos_PBC(float pos, float l)
	// This function calculates and returns the positions of the 
	// atoms with periodic boundary conditions used.  
{
	float pos_PBC;
	if (pos < (0.0))
		pos_PBC = pos + l;
	else if (pos > (l)) 
		pos_PBC = pos - l;
	else
		pos_PBC = pos;
	return pos_PBC;
}

__device__ float separation_PBC(float ds, float l)
{
	float s_PBC;
	if (ds > (0.5*l)) 
		s_PBC = ds - l;
	else if (ds < (-0.5*l)) 
		s_PBC = ds + l;
	else
		s_PBC = ds;
	return s_PBC;
}

__device__ int getGlobalIdx_3D_3D_l(int l)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z * 4) + (threadIdx.z * (blockDim.x * blockDim.y * 4)) + (threadIdx.y * blockDim.x * 4 )+ threadIdx.x * 4 + l;
	return threadId;
}

__device__ int getGlobalIdx_3D_1D() {
int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * blockDim.x + threadIdx.x;
return threadId;
}

__device__ int getGlobalIdx_3D_3D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z ) + (threadIdx.z * (blockDim.x * blockDim.y )) + (threadIdx.y * blockDim.x )+ threadIdx.x ;
	return threadId;
}

__device__ int getGlobalIdx_1D_1D()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int check_position (float3 *d_gr_centerp, float x, float y, float z, int * grain)
	// This function checks to see if the atom's position is
	// closer to the center of the current grain than it is to 
	// any other grain. IF so, check is assigned 1. If not, check 
	// is assigned 0.
{
	int i, check;
	float r12,r22,dx,dy,dz;

	check = 1;

	//check if atom is outside the outer periodic image cells
	if (x >= 2.0*d_lx || x <= -d_lx) 
		check = 0;
	else if (y >= 2.0*d_ly || y <= -d_ly)
		check = 0;
	else if (z >= 2.0*d_lz || z <= -d_lz) 
		check = 0;
	if (check == 0) return 0;

	dx = d_gr_centerp[*grain].x - x;
	dy = d_gr_centerp[*grain].y - y;
	dz = d_gr_centerp[*grain].z - z;

	//check if atom is nearest to the actual grain center (and not it's image)
	if (abs(dx) > 0.5*d_lx) 
		check = 0;
	else if (abs(dy) > 0.5*d_ly) 
		check = 0;
	else if (abs(dz) > 0.5*d_lz) 
		check = 0;
	if (check == 0) return 0;

	//check if atom is closest to current grain center 
	r12 = dx*dx+dy*dy+dz*dz;
	for ( i = 0; i < d_numgrain; ++i)
	{
		if (i == *grain) continue;
		dx = d_gr_centerp[i].x - x;
		dy = d_gr_centerp[i].y - y;
		dz = d_gr_centerp[i].z - z;
		dx = separation_PBC(dx,d_lx);
		dy = separation_PBC(dy,d_ly);
		dz = separation_PBC(dz,d_lz);
		r22 = dx*dx+dy*dy+dz*dz;
		if (r22 <= r12) 
		{	check = 0;
		break;
		}
	}

	return check;
}


// Heavy calculation
__global__ void assign_initial_positions(float3 *d_gr_centerp, float3 *d_r, float *d_alpha, float *d_beta, float *d_gama, bool *d_atom_id,int *d_grain, int *d_l1)
{
	int check,l1;
	int i = d_nx[0] + threadIdx.x + blockIdx.x * blockDim.x;
	int j = d_ny[0] + threadIdx.y + blockIdx.y * blockDim.y;
	int k = d_nz[0] + threadIdx.z + blockIdx.z * blockDim.z;
	int n1;
	float x1,y1,z1,x_rot,y_rot,z_rot;
    float h11,h12,h13;
    float h21,h22,h23;
    float h31,h32,h33;
	float basis[4][3];
	float phi1, phi2, phi3;


	// Distribution threads here
	if ( i < d_nx[1] && j < d_ny[1] && k < d_nz[1] )
	{
		basis[0][0]=0.00;
		basis[0][1]=0.00;
		basis[0][2]=0.00;
		basis[1][0]=0.50;
		basis[1][1]=0.50;
		basis[1][2]=0.00;
		basis[2][0]=0.00;
		basis[2][1]=0.50;
		basis[2][2]=0.50;
		basis[3][0]=0.50;
		basis[3][1]=0.00;
		basis[3][2]=0.50;

		// Tilt the grains.
		phi1 = d_alpha[*d_grain]*M_PI/180.0;
		phi2 = d_beta[*d_grain]*M_PI/180.0;
		phi3 = d_gama[*d_grain]*M_PI/180.0;

	    h11=cos(phi1)*cos(phi3)-sin(phi1)*sin(phi3)*cos(phi2);
	    h12=sin(phi1)*cos(phi3)+cos(phi1)*sin(phi3)*cos(phi2);
	    h13=sin(phi3)*sin(phi2);
	    h21=-(cos(phi1)*sin(phi3)+sin(phi1)*cos(phi3)*cos(phi2));
	    h22=-sin(phi1)*sin(phi3)+cos(phi1)*cos(phi3)*cos(phi2);
	    h23=cos(phi3)*sin(phi2);
	    h31=sin(phi1)*sin(phi2);
	    h32=-cos(phi1)*sin(phi2);
	    h33=cos(phi2);

			l1=*d_l1;
			n1 = getGlobalIdx_3D_3D();
			x1 = i*d_a + basis[l1][0]*d_a;
			y1 = j*d_a + basis[l1][1]*d_a;
			z1 = k*d_a + basis[l1][2]*d_a;
			x_rot = (x1*h11 + y1*h21 + z1*h31)+ d_gr_centerp[*d_grain].x;
			y_rot = (x1*h12 + y1*h22 + z1*h32)+ d_gr_centerp[*d_grain].y;
			z_rot = (x1*h13 + y1*h23 + z1*h33)+ d_gr_centerp[*d_grain].z;

			check = check_position(d_gr_centerp, x_rot,y_rot,z_rot,d_grain);
			if (check == 1)	
			{
				d_r[n1].x = pos_PBC(x_rot,d_lx);
				d_r[n1].y = pos_PBC(y_rot,d_ly);
				d_r[n1].z = pos_PBC(z_rot,d_lz);
				d_atom_id[n1]  = 1;
			}

	}
	//__syncthreads();


}

__global__ void get_GBlist(float3 *d_gr_centerp, float3 *d_r, int *d_atom_grain,int *d_atom_neigh, bool *d_tag)
{
	int i, j, mygrain;
	float dx1, dx2, dx, dy1, dy2, dy, dz1, dz2, dz, r12, r22, r32, r1, r3;
	float co, projec, dis;
	float d_mini;
	i = getGlobalIdx_3D_3D();
	if ( i < d_natom)
	{
		mygrain = d_atom_grain[i];
		dx1 = separation_PBC(d_r[i].x - d_gr_centerp[mygrain].x,d_lx);
		dy1 = separation_PBC(d_r[i].y - d_gr_centerp[mygrain].y,d_ly);
		dz1 = separation_PBC(d_r[i].z - d_gr_centerp[mygrain].z,d_lz);
		r12 = dx1*dx1+dy1*dy1+dz1*dz1;
		r1 = sqrt(r12);
		d_mini = d_a;
		for ( j = 0; j < d_numgrain; ++j)
		{
			if ( j == mygrain ) continue;
			dx = separation_PBC(d_r[i].x - d_gr_centerp[j].x,d_lx);
			dy = separation_PBC(d_r[i].y - d_gr_centerp[j].y,d_ly);
			dz = separation_PBC(d_r[i].z - d_gr_centerp[j].z,d_lz);
			r22 = dx*dx+dy*dy+dz*dz;
			dx2 = separation_PBC(d_gr_centerp[mygrain].x - d_gr_centerp[j].x,d_lx);
			dy2 = separation_PBC(d_gr_centerp[mygrain].y - d_gr_centerp[j].y,d_ly);
			dz2 = separation_PBC(d_gr_centerp[mygrain].z - d_gr_centerp[j].z,d_lz);
			r32 = dx2*dx2+dy2*dy2+dz2*dz2;
			r3 = sqrt(r32);
			// What's this?
			co = (r12+r32-r22)/2.0/r1/r3;
			projec = r1*co;
			dis = r3/2.0 - projec;
			if (i == 0 && j == 0)
				printf("d_natom = %d, dis = %f\n",d_natom,dis);
//			if (dis < 0.22*d_a)
			if (dis < 0.27*d_a)
			{			
				d_tag[i] = 1;
				if (dis < d_mini)
				{
					d_mini = dis;
					d_atom_neigh[i] = j;
				}
			}
		}
	}
}

__global__ void clean_grain_boundaries(float3 *d_r, int *d_atom_grain,int *d_atom_neigh, int *d_GBlist, bool *d_tag)
{

	int i, j, ii, jj;
	float dx, dy, dz, dr2;
	float a2 = d_a*d_a;
	ii = getGlobalIdx_3D_1D();
//	d_tag[d_GBlist[getGlobalIdx_1D_1D()]] = getGlobalIdx_1D_1D();
//	if ( ii < d_GBatoms/10 && ii > d_GBatoms/100)
		if (ii < d_GBatoms)
	{	
		i = d_GBlist[ii];
		for ( jj = ii+1; jj < d_GBatoms; ++jj)
		{
			j = d_GBlist[jj];
			if (d_atom_neigh[i] != d_atom_grain[j] || d_atom_neigh[j] != d_atom_grain[i])
				continue;
			dx = d_r[i].x - d_r[j].x;
			dy = d_r[i].y - d_r[j].y;
			dz = d_r[i].z - d_r[j].z;
			dx = separation_PBC(dx,d_lx);
			dy = separation_PBC(dy,d_ly);
			dz = separation_PBC(dz,d_lz);
			dr2 = dx*dx+dy*dy+dz*dz;
//				if (dr2 <= 0.17*a2)
			if (dr2 <= 0.215*a2)
			{
				d_tag[i] = 1;
//				d_drlist[ii] = dr2;
				break;
			}
		}
	}

}

void  create_sample()
{
	int i, grain,l;
	int nx[2],ny[2],nz[2];
	int aindex;
	float3 *d_gr_centerp, *d_r;
	float3 *rr;
//	float *d_atom_mini;

	int *d_atom_grain, *d_atom_neigh, *d_grain, *d_l1; 
	float *d_alpha ,*d_beta ,*d_gama;
	bool *d_atom_id;
	nx[0] = int(-0.88*(lx/a));
	nx[1] = int(0.88*(lx/a));
	ny[0] = int(-0.88*(ly/a)*ratio);
	ny[1] = int(0.88*(ly/a)*ratio);
	nz[0] = int(-0.88*(lz/a)*ratio);
	nz[1] = int(0.88*(lz/a)*ratio);
	DIM.x = nx[1]-nx[0]+1;
	DIM.y = ny[1]-ny[0]+1;
	DIM.z = nz[1]-nz[0]+1;
	printf("DIM.x = %d, DIM.y = %d, DIM.z = %d\n", DIM.x, DIM.y, DIM.z);
	if (DIM.x <10) DIM.x = 10;
	if (DIM.y <10) DIM.y = 10;
	if (DIM.z <10) DIM.z = 10;

	nmax = DIM.x * DIM.y * DIM.z;
	printf("Maximum atom number: %d\n", nmax);
	r = new float3[nmax/5*4];
	rr = new float3[nmax];
	atom_id = new bool[nmax];
	memset(atom_id, 0, nmax*sizeof(bool));
	atom_grain = new int[nmax/5*4];
//	atom_neigh = new int[nmax];
//	atom_mini = new float[nmax];
	

	clock_t begin = clock();
	// CUDA memoray allocation
	hipMalloc(&d_r, nmax*sizeof(float3));
	hipMalloc(&d_atom_id, nmax*sizeof(bool));
	hipMalloc(&d_grain, 1*sizeof(int));
	hipMalloc(&d_gr_centerp, numgrain*sizeof(float3));
	hipMalloc(&d_alpha, numgrain*sizeof(float));
	hipMalloc(&d_beta, numgrain*sizeof(float));
	hipMalloc(&d_gama, numgrain*sizeof(float));
	hipMalloc(&d_l1, 1*sizeof(int));




	// Device initiallization
	hipMemcpyToSymbol(HIP_SYMBOL(d_lx), &lx, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ly), &ly, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_lz), &lz, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_numgrain), &numgrain, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nmax), &nmax, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_a), &a, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nx), nx, 2*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ny), ny, 2*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nz), nz, 2*sizeof(int));
	hipMemcpy(d_gr_centerp, gr_centerp, numgrain*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(d_alpha, alpha, numgrain*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_beta, beta, numgrain*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gama, gama, numgrain*sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy(d_atom_id, atom_id, nmax*sizeof(bool), hipMemcpyHostToDevice);


	//if (DIM.z > 640)
	//{
	//	printf("Warning: DIM.z is larger than 640, set DIM.z to 640");
	//	DIM.z = 640;
	//}
	dim3 blocks((DIM.x+8-1)/8, (DIM.y+8-1)/8, (DIM.z+8-1)/8);
	dim3 threads(8, 8, 8);

	// Initlal positions
	for ( grain=0; grain < numgrain; ++grain )
	{
		printf ("%d\n",grain);
		hipMemcpy(d_grain, &grain, 1*sizeof(int), hipMemcpyHostToDevice);
		//CUDA//
	for (l=0;l<4;++l){
		hipMemset(d_atom_id, 0, nmax*sizeof(bool));
		hipMemcpy(d_l1, &l, 1*sizeof(int), hipMemcpyHostToDevice);
		assign_initial_positions<<< blocks, threads >>>(d_gr_centerp, d_r, d_alpha, d_beta, d_gama, d_atom_id, d_grain,d_l1);
		//CUDA END//
		hipMemcpy(rr, d_r, nmax*sizeof(float3), hipMemcpyDeviceToHost);
		hipMemcpy(atom_id, d_atom_id, nmax*sizeof(bool), hipMemcpyDeviceToHost);
		for (i =0; i < nmax; ++i)
		{
			if (atom_id[i] == 1)	
			{
				count = count+1;
				aindex = count-1;
				r[aindex].x = rr[i].x;
				r[aindex].y = rr[i].y;
				r[aindex].z = rr[i].z;
				atom_grain[aindex] = grain;			
			}
		}
		}
	}
	free(rr);



	natom = count;
	printf ("Initial atom number: %d\n",natom);
	
	clock_t end = clock();
	float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
	cout << "Time elapsed: " << elapsed_secs << " s" << endl;
	
	clock_t begin1 = clock();
	//Clean grain boundaries 
	hipFree(d_r);
	hipFree(d_atom_id);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(d_gama);

	int * d_GBlist, * GBlist;
	int counter2,counter3;
	bool *d_tag, *tag;
//	float * d_drlist;

	int GBatoms = 0;
	GBlist = new int[natom];
	tag = new bool[natom];
	atom_neigh = new int[natom];
//	atom_mini = new float[natom];


	hipMalloc(&d_r, natom*sizeof(float3));
	hipMalloc(&d_atom_grain, natom*sizeof(int));
	hipMalloc(&d_atom_neigh, natom*sizeof(int));
//	hipMalloc(&d_atom_mini, natom*sizeof(float));
	hipMalloc(&d_tag, natom*sizeof(bool));

	hipMemset(d_tag, 0, natom*sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(d_natom), &natom, sizeof(int));
	hipMemcpy(d_r, r, natom*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(d_atom_grain, atom_grain, natom*sizeof(int), hipMemcpyHostToDevice);

	dim3 blocks2((natom+32768-1)/32768, 8, 8);
	dim3 threads2(8, 8, 8);

	get_GBlist <<< blocks2, threads2 >>> (d_gr_centerp, d_r, d_atom_grain, d_atom_neigh, d_tag);
	hipMemcpy(tag, d_tag, natom*sizeof(bool), hipMemcpyDeviceToHost);
	for ( i = 0; i < natom; ++i)
	{
if (tag[i] == 1)
		{
			++GBatoms;
			GBlist[GBatoms-1] = i;
	//cout << i << " " << tag[i] << endl;
		}
	}
	printf ("GBatoms: %d\n",GBatoms);

	clock_t end1 = clock();
	float elapsed_secs1 = float(end1 - begin1) / CLOCKS_PER_SEC;
	cout << "Time elapsed: " << elapsed_secs1 << " s" << endl;
	
//	drlist = new float[GBatoms];
	clock_t begin2 = clock();

//	hipFree(d_atom_mini);
	hipFree(d_gr_centerp);

	hipMemset(d_tag, 0, natom*sizeof(bool));
	hipMalloc(&d_GBlist, GBatoms*sizeof(int));
	hipMemcpy(d_GBlist, GBlist, GBatoms*sizeof(int), hipMemcpyHostToDevice);

//	hipMalloc(&d_drlist, GBatoms*sizeof(float));
//	hipMemset(d_drlist, 100 , GBatoms*sizeof(float));
	for (i=0;i<GBatoms;++i){tag[GBlist[i]]=0;}
//	hipMemcpy(d_drlist, drlist, GBatoms*sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy(drlist, d_drlist, GBatoms*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(GBlist, d_GBlist, GBatoms*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpyToSymbol(HIP_SYMBOL(d_GBatoms), &GBatoms, sizeof(int));
//	hipMemcpy(tag, d_tag, natom*sizeof(int), hipMemcpyDeviceToHost);
	printf("%d %d\n",natom,GBatoms);
//	for (i=0;i<GBatoms;++i){cout << tag[GBlist[i]] << " " << drlist[i] << " " << GBlist[i] << endl;}
	dim3 blocks3((GBatoms+16384-1)/16384,8,8);
	dim3 threads3(256);
	clean_grain_boundaries <<< blocks3, threads3 >>> (d_r, d_atom_grain, d_atom_neigh, d_GBlist, d_tag);
//hipError_t error = hipGetLastError();
//printf("CUDA error: %s\n", hipGetErrorString(error));
	counter2 = -1;
	counter3 = 0;
//	hipMemcpy(drlist, d_drlist, GBatoms*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(tag, d_tag, natom*sizeof(bool), hipMemcpyDeviceToHost);
//	for (i=0;i<GBatoms;++i){cout << tag[GBlist[i]] << " " << drlist[i] << " " << GBlist[i] << endl;}

//	cout << natom << endl;
	for (i = 0; i < natom; ++i)
	{
		if (tag[i] == 0) 
		{
			counter2 = counter2 + 1;
			r[counter2].x = r[i].x;
			r[counter2].y = r[i].y;
			r[counter2].z = r[i].z;
//			cout << i << " " << tag[i] << " " << counter2 << endl;
		}
//			else {counter3++;cout << i << " " << tag[i] << " " << counter2 << endl;}
				else {counter3++;}
	}
	natom = counter2+1;
	printf ("Atom number (after cleaning): %d %d\n",natom,counter3);
	
	clock_t end2 = clock();
	float elapsed_secs2 = float(end2 - begin2) / CLOCKS_PER_SEC;
	cout << "Time elapsed: " << elapsed_secs2 << " s" << endl;

}

int write_output_files(char* ofn)
{
	int i;
	ofstream ofile;
	ofile.open(ofn);
	ofile << "# Position data for Cu system" << endl;
	ofile << "" << endl;
	ofile << natom << " atoms" << endl;
	ofile << "1 atom types" << endl;
	ofile << "" << endl;
	ofile << "0 " << lx << " xlo xhi" << endl;
	ofile << "0 " << ly << " ylo yhi" << endl;
	ofile << "0 " << lz << " zlo zhi" << endl;
	ofile << "" << endl;
	ofile << "Masses" << endl;
	ofile << "" << endl;
	ofile << "1 63.55" << endl;
	ofile << "" << endl;
	ofile << "Atoms" << endl;
	ofile << "" << endl;
	for ( i = 0; i < natom; ++i)
	{
		ofile << i+1 << " 1 ";
		ofile << setprecision(6) << r[i].x <<  " " << r[i].y << " " << r[i].z << endl;
	}
	ofile.close();
	return 0;
}

void write_output_cfg(char* ofn)
{
	int i;
	ofstream ofile;
	ofile.open(ofn);
	ofile << "Number of particles = " << natom << endl;
	ofile << "A = 1 Angstrom (basic length-scale)" << endl;
	ofile << "H0(1,1) = " << lx << " A" << endl;
	ofile << "H0(1,2) = 0 A" << endl;
	ofile << "H0(1,3) = 0 A" << endl;
	ofile << "H0(2,1) = 0 A" << endl;
	ofile << "H0(2,2) = " << ly << " A" << endl;
	ofile << "H0(2,3) = 0 A" << endl;
	ofile << "H0(3,1) = 0 A" << endl;
	ofile << "H0(3,2) = 0 A" << endl;
	ofile << "H0(3,3) = " << lz << " A" << endl;
	ofile << ".NO_VELOCITY." << endl;
	ofile << "entry_count = 3" << endl;
	ofile << "63.55" << endl;
	ofile << "Cu" << endl;
	for ( i = 0; i < natom; ++i)
	{
		ofile << setprecision(5) << r[i].x/lx <<  " " << r[i].y/ly << " " << r[i].z/lz << endl;
	}
	ofile.close();
}

int main(int argc, char* argv[])
{
	int deviceCount;

	hipGetDeviceCount(&deviceCount);
	printf("Number of GPU devices: %d\n", deviceCount);


	clock_t begin3 = clock();


	char* ofn; //output filename
	char* ifn; //input filename
	if (argc < 3 || strncmp(argv[1],"-h",2) == 0 || strncmp(argv[1],"--help",6) == 0)
	{cout << "./ggp input output" << endl;cout << "Example: ./ggp input.txt a.out" << endl;return 0;}
	ifn = argv[1];
	ofn = argv[2];
	read_config(ifn);
	create_sample();
	
	clock_t end3 = clock();
	
	float elapsed_secs3 = float(end3 - begin3) / CLOCKS_PER_SEC;

	cout << "Total time elapsed: " << elapsed_secs3 << " s" << endl;
	
	cout << "Writing file..." << endl;
//	write_output_files(ofn);
	write_output_cfg(ofn);
	cout << "Done" << endl;
	
	

	
	return 0;

}

